
#include <hip/hip_runtime.h>
#include <iostream>



__global__ void factorial(const int N, int *f){
  if ( N >= 1) {
    *f = *f*N;
    factorial<<<1,1>>>(N-1,f);
  }
}

using namespace std;

int main()
{

    int N = 5;
    int *f;
    hipMallocManaged(&f,sizeof(int));
    *f = 1;
    factorial<<<1,1>>>(N,f);
    hipDeviceSynchronize();
    cout << *f << "\n";

    return EXIT_SUCCESS;
}
